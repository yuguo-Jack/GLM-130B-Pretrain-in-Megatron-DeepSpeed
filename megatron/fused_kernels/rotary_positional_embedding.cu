#include "hip/hip_runtime.h"

/* coding=utf-8
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cfloat>
#include <limits>
#include <stdint.h>
#include <c10/macros/Macros.h>
#include <ATen/ATen.h>
#include <ATen/Utils.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
// #include <hip/hip_runtime_api.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include "type_shim.h"

#define A(i, j, k, r) A[i * b * np * hn + j * np * hn + k * hn + r]
#define B(i, j, r) B[i * b * hn + j * hn + r]
#define C(i, j, r) C[i * b * hn + j * hn + r]
#define res(i, j, k, r) res[i * b * np * hn + j * np * hn + k * hn + r]
#define res_grad(i, j, k, r) \
    res_grad[i * b * np * hn + j * np * hn + k * hn + r]
#define grad_out(i, j, k, r) \
    grad_out[i * b * np * hn + j * np * hn + k * hn + r]

const int block_np = 16, block_hn = 16;

template <typename U>
__global__ __global__ void tensorMult_gpu(int sq, int b, int np, int hn,
                                          const U* A, const U* B, const U* C,
                                          U* res) {
    int i = blockIdx.x, j = blockIdx.y;
    int k0 = threadIdx.y, r0 = threadIdx.x;
    for (int k = k0; k < np; k += 16) {
        for (int r = r0; r < hn; r += 16) {
            U temp = (r + hn / 2 < hn) ? -A(i, j, k, r + hn / 2)
                                       : A(i, j, k, r + hn / 2 - hn);
            res(i, j, k, r) = A(i, j, k, r) * B(i, j, r) + temp * C(i, j, r);
        }
    }
}

template <typename U>
void host_apply_tensorMult_gpu(int sq, int b, int np, int hn, const U* A,
                               const U* B, const U* C, U* res) {
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    dim3 blocks(sq, b);
    dim3 threads(block_hn, block_np);
    tensorMult_gpu<<<blocks, threads, 0, stream>>>(sq, b, np, hn, A,
                                                              B, C, res);
}

void tensorMult_gpu_launch(int sq, int b, int np, int hn, at::Tensor* A,
                           at::Tensor* B, at::Tensor* C, at::Tensor* res) {
    DISPATCH_FLOAT_HALF_AND_BFLOAT_TYPES(
        B->scalar_type(), "forward_gpu",
        host_apply_tensorMult_gpu(
            sq, b, np, hn, A->data_ptr<scalar_t>(), B->data_ptr<scalar_t>(),
            C->data_ptr<scalar_t>(), res->data_ptr<scalar_t>());)
}

template <typename U>
__global__ __global__ void tensorMult_backward_gpu(int sq, int b, int np,
                                                   int hn, const U* grad_out,
                                                   const U* B, const U* C,
                                                   U* res_grad) {
    int i = blockIdx.x, j = blockIdx.y;
    int k0 = threadIdx.y, r0 = threadIdx.x;
    for (int k = k0; k < np; k += 16) {
        for (int r = r0; r < hn; r += 16) {
            U temp1 = (r + hn / 2 < hn) ? grad_out(i, j, k, r + hn / 2)
                                        : grad_out(i, j, k, r + hn / 2 - hn);
            U temp2 = (r + hn / 2 < hn) ? C(i, j, r + hn / 2)
                                        : -C(i, j, r + hn / 2 - hn);
            res_grad(i, j, k, r) =
                grad_out(i, j, k, r) * B(i, j, r) + temp1 * temp2;
        }
    }
}

template <typename U>
void host_apply_tensorMult_backward_gpu(int sq, int b, int np, int hn,
                                        const U* grad_out, const U* B,
                                        const U* C, U* res_grad) {
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    dim3 blocks(sq, b);
    dim3 threads(block_hn, block_np);
    tensorMult_backward_gpu<<<blocks, threads, 0, stream>>>(
        sq, b, np, hn, grad_out, B, C, res_grad);
}

void tensorMult_backward_gpu_launch(int sq, int b, int np, int hn,
                                    at::Tensor* grad_out, at::Tensor* B,
                                    at::Tensor* C, at::Tensor* res_grad) {
    DISPATCH_FLOAT_HALF_AND_BFLOAT_TYPES(
        B->scalar_type(), "backward_gpu",
        host_apply_tensorMult_backward_gpu(
            sq, b, np, hn, grad_out->data_ptr<scalar_t>(),
            B->data_ptr<scalar_t>(), C->data_ptr<scalar_t>(),
            res_grad->data_ptr<scalar_t>());)
}
